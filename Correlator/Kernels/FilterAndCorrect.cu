#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>
#include <hip/hip_math_constants.h>
//#include <math_functions.h>
//#include <cstdio>
#include <cufftdx.hpp>
#if defined __CUDA_ARCH__
#include <sm_61_intrinsics.hpp> // must include this explicitly for __dp4a when compiling with NVRTC
#endif


//#define SUBBAND_BANDWIDTH 195312.5

#define REAL	0
#define IMAG	1
#define COMPLEX	2


typedef float2 fcomplex; // std::complex not properly supported yet

__device__ inline fcomplex operator + (fcomplex a, fcomplex b)
{
  return make_float2(a.x + b.x, a.y + b.y);
}

__device__ inline fcomplex operator - (fcomplex a, fcomplex b)
{
  return make_float2(a.x - b.x, a.y - b.y);
}

__device__ inline fcomplex operator * (float a, fcomplex b)
{
  return make_float2(a * b.x, a * b.y);
}

#if 0
__device__ inline __half2 operator * (__half2 a, __half2 b)
{
  return make_half2(a.x * b.x - a.y * b.y, a.x * b.y - a.y * b.x); // TODO: use vector intrinsics
}
#endif

__device__ inline fcomplex operator += (fcomplex &a, fcomplex b)
{
  return make_float2(a.x += b.x, a.y += b.y);
}


#define complexMul(a,b) make_float2((a).x * (b).x - (a).y * (b).y, (a).y * (b).x + (a).x * (b).y)


#if NR_BITS == 16
typedef short2  InputSample;
typedef __half2 OutputSample;
#elif NR_BITS == 8
typedef char2 InputSample, OutputSample;
#else
#error unsupport NR_BITS
#endif


template <typename T> __device__ inline __half2 sampleToComplexFloat(T sample)
{
  return make_half2(sample.x, sample.y);
}


#define NR_TIMES_PER_BLOCK      	(128 / (NR_BITS))
#define CHANNEL_INTEGRATION_FACTOR	(NR_CHANNELS_PER_SUBBAND == 1 ? 1 : (NR_CHANNELS_PER_SUBBAND - 1) / NR_OUTPUT_CHANNELS_PER_SUBBAND)
#define NR_OUTPUT_SAMPLES_PER_CHANNEL	(CHANNEL_INTEGRATION_FACTOR * NR_SAMPLES_PER_CHANNEL)


__device__ inline bool time_ok(unsigned time)
{
  return (NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS_PER_SUBBAND % 64 == 0 || time < (NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS_PER_SUBBAND;
}


__device__ inline bool recv_pol_ok(unsigned recv_pol)
{
  return NR_RECEIVERS * NR_POLARIZATIONS % 64 == 0 || recv_pol < NR_RECEIVERS * NR_POLARIZATIONS;
}


__device__ inline bool output_time_ok(unsigned time)
{
  return NR_SAMPLES_PER_CHANNEL % 64 == 0 || time < NR_SAMPLES_PER_CHANNEL;
}


__device__ inline bool output_channel_ok(unsigned channel)
{
  return NR_OUTPUT_CHANNELS_PER_SUBBAND * CHANNEL_INTEGRATION_FACTOR % 64 == 0 || channel < NR_OUTPUT_CHANNELS_PER_SUBBAND * CHANNEL_INTEGRATION_FACTOR;
}


extern "C" __global__ __launch_bounds__(32 * 32)
void transpose(
  InputSample output[NR_RECEIVERS * NR_POLARIZATIONS][(NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS_PER_SUBBAND],
#if defined HAS_INTEGRATED_MEMORY && !defined TEST
  const InputSample input[NR_RING_BUFFER_SAMPLES_PER_SUBBAND][NR_RECEIVERS * NR_POLARIZATIONS],
  unsigned startIndex
#else
  const InputSample input[(NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS_PER_SUBBAND][NR_RECEIVERS * NR_POLARIZATIONS]
#endif
)
{
  __shared__ InputSample tmp[64][64 + 1 /* one wider, to avoid bank conflicts */];

  unsigned recv_pol_major = blockIdx.x * 64;
  unsigned time_major     = blockIdx.y * 64;

  for (unsigned y = 0; y < 64; y += 32) {
    for (unsigned x = 0; x < 64; x += 32) {
      unsigned time     = time_major     + y + threadIdx.y;
      unsigned recv_pol = recv_pol_major + x + threadIdx.x;

      if (time_ok(time) && recv_pol_ok(recv_pol))
#if defined HAS_INTEGRATED_MEMORY && !defined TEST
	tmp[y + threadIdx.y][x + threadIdx.x] = input[(time + startIndex) % NR_RING_BUFFER_SAMPLES_PER_SUBBAND][recv_pol];
#else
	tmp[y + threadIdx.y][x + threadIdx.x] = input[time][recv_pol];
#endif
    }
  }

  __syncthreads();

  for (unsigned y = 0; y < 64; y += 32) {
    for (unsigned x = 0; x < 64; x += 32) {
      unsigned time     = time_major     + x + threadIdx.x;
      unsigned recv_pol = recv_pol_major + y + threadIdx.y;

      if (time_ok(time) && recv_pol_ok(recv_pol))
	output[recv_pol][time] = tmp[x + threadIdx.x][y + threadIdx.y];
    }
  }
}


extern "C" __global__ __launch_bounds__(NR_CHANNELS_PER_SUBBAND)
void filterAndCorrect(
  //OutputSample      output[NR_OUTPUT_CHANNELS_PER_SUBBAND][NR_OUTPUT_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK][NR_RECEIVERS][NR_POLARIZATIONS][NR_TIMES_PER_BLOCK],
  OutputSample      output[NR_RECEIVERS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_OUTPUT_CHANNELS_PER_SUBBAND * CHANNEL_INTEGRATION_FACTOR],
  const InputSample input[NR_RECEIVERS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1][NR_CHANNELS_PER_SUBBAND],
  const float       filterWeights[NR_CHANNELS_PER_SUBBAND][NR_TAPS],
  const float       delays[NR_RECEIVERS][NR_POLARIZATIONS], // in seconds
  const float       bandPassWeights[NR_CHANNELS_PER_SUBBAND]
  //double	    subbandFrequency
)
{
#if defined __CUDA_ARCH__
  //unsigned infinity_count = 0, total_count = 0;

  //unsigned channel      = threadIdx.x;
  unsigned tid          = NR_CHANNELS_PER_SUBBAND / 16 * threadIdx.y + threadIdx.x;
  unsigned channel      = tid;
  unsigned polarization = blockIdx.x;
  unsigned receiver     = blockIdx.y;

  using namespace cufftdx;
  using FFT = decltype(Block() +
	      Size<NR_CHANNELS_PER_SUBBAND>() +
	      Type<fft_type::c2c>() +
	      Direction<fft_direction::forward>() +
	      Precision<float>() +
	      ElementsPerThread<16>() +
	      FFTsPerBlock<16>() +
	      //BlockDim<NR_CHANNELS_PER_SUBBAND, 1, 1>() + // Not yet supported by cuFFTDx
	      SM<__CUDA_ARCH__>());

  __shared__ FFT::value_type shared_mem[FFT::shared_memory_size / sizeof(FFT::value_type)];

#if NR_BITS == 16
  __half2 history[NR_TAPS], _weights[NR_TAPS];
#elif NR_BITS == 8
  union {
    signed char ch[NR_TAPS];
    char4       ch4[NR_TAPS / 4 + 1];
    int         i[NR_TAPS / 4 + 1];
  } history[COMPLEX], _weights;
#endif

  for (unsigned time = 0; time < NR_TAPS - 1; time ++) {
#if NR_BITS == 16
    history[time + 1] = sampleToComplexFloat(input[receiver][polarization][time][channel]);
#elif NR_BITS == 8
    history[REAL].ch[time + 1] = input[receiver][polarization][time][channel].x;
    history[IMAG].ch[time + 1] = input[receiver][polarization][time][channel].y;
#endif
  }

  for (unsigned tap = 0; tap < NR_TAPS; tap ++)
#if NR_BITS == 16
    //_weights[tap] = __float2half2_rn(filterWeights[channel][NR_TAPS - 1 - tap]);
    _weights[tap] = __float2half2_rn(filterWeights[channel][tap] / 16);
#elif NR_BITS == 
    //_weights.ch[tap] = roundf(128 * filterWeights[channel][NR_TAPS - 1 - tap]);
    _weights.ch[tap] = roundf(127 * filterWeights[channel][tap]);
#endif

  __shared__ float2 fft_data[16][NR_CHANNELS_PER_SUBBAND];

  //float  phi = -2 * HIP_PI_F * delays[receiver][polarization];
  //float2 v   = make_float2(cosf(phi), sinf(phi));

  for (unsigned time_major = 0; time_major < NR_SAMPLES_PER_CHANNEL; time_major += 16) {
#pragma unroll
    for (unsigned time_minor = 0; time_minor < 16; time_minor ++) {
#if NR_BITS == 16
      history[time_minor] = sampleToComplexFloat(input[receiver][polarization][time_major + time_minor + NR_TAPS - 1][channel]);

      __half2 sum = make_half2(0, 0);

#pragma unroll
      for (unsigned tap = 0; tap < NR_TAPS; tap ++)
	sum = __hfma2(_weights[tap], history[(time_minor - tap) % NR_TAPS], sum);

      fft_data[time_minor][channel] = __half22float2(sum);
#elif NR_BITS == 8
      char2 sample = input[receiver][polarization][time_major + time_minor + NR_TAPS - 1][channel];
      history[REAL].ch[0] = sample.x;
      history[IMAG].ch[0] = sample.y;

      int sum[COMPLEX] = {0};

      for (unsigned ri = 0; ri < COMPLEX; ri ++) {
#if 0
	for (int tap = NR_TAPS / 4; -- tap >= 0;) {
	  sum[ri] = __dp4a(history[ri].i[tap], _weights.i[tap], sum[ri]);
	  history[ri].i[tap] = __funnelshift_l(history[ri].i[tap], history[ri].i[tap - 1], 8);
	}
#else
	for (unsigned tap = 0; tap < NR_TAPS; tap ++)
	  sum[ri] += (int) history[ri].ch[tap] * _weights.ch[tap];

	for (unsigned tap = NR_TAPS - 1; tap > 0; tap --)
	  history[ri].ch[tap] = history[ri].ch[tap - 1];
#endif
      }

      fft_data[time_minor][channel] = make_float2(sum[REAL], sum[IMAG]);
#endif
    }

    __syncthreads();

    FFT::value_type thread_data[FFT::storage_size];
    using complex_type = typename FFT::value_type;

    unsigned stride = size_of<FFT>::value / FFT::elements_per_thread;

    for (unsigned i = 0; i < FFT::elements_per_thread; ++ i)
      thread_data[i] = * ((FFT::value_type *) &fft_data[threadIdx.y][i * stride + threadIdx.x]);

    FFT().execute(thread_data, shared_mem);

    for (unsigned i = 0; i < FFT::elements_per_thread; ++ i)
      * ((complex_type *) &fft_data[threadIdx.y][i * stride + threadIdx.x]) = thread_data[i];

    __syncthreads();

    {
#if 0
      unsigned time_minor    = tid % 16U;
      unsigned channel_minor = tid / 16U;

      for (unsigned channel_major = 0; channel_major < NR_CHANNELS_PER_SUBBAND; channel_major += NR_CHANNELS_PER_SUBBAND / 16) {
	unsigned channel = channel_major + channel_minor;
	unsigned output_channel = (channel - 1) / CHANNEL_INTEGRATION_FACTOR;
	unsigned time = time_major + time_minor;
	unsigned output_time = time + ((channel - 1) % CHANNEL_INTEGRATION_FACTOR) * NR_SAMPLES_PER_CHANNEL;

	if (output_channel < NR_OUTPUT_CHANNELS_PER_SUBBAND) {
	  //double frequency = subbandFrequency - .5 * SUBBAND_BANDWIDTH + channel * (SUBBAND_BANDWIDTH / NR_CHANNELS_PER_SUBBAND);
	  float2 sample = bandPassWeights[channel] * fft_data[time_minor][channel]; // TODO: use __half2 ???
	  //sample = complexMul(sample, v);
//if ((float) sample.x != 0 || (float) sample.y != 0)
  //printf("GPU: output[%u][%u][%u][%u][%u] = (%f,%f)\n", output_channel, output_time / NR_TIMES_PER_BLOCK, receiver, polarization, output_time % NR_TIMES_PER_BLOCK, (float) sample.x, (float) sample.y);
#if NR_BITS == 16
	  //if (__hisinf(__habs(__float2half_rn(sample.x))))
            //++ infinity_count;
          //if (__hisinf(__habs(__float2half_rn(sample.y))))
            //++ infinity_count;
	  //total_count += 2;

	  output[output_channel][output_time / NR_TIMES_PER_BLOCK][receiver][polarization][output_time % NR_TIMES_PER_BLOCK] = __float22half2_rn(sample);
#elif NR_BITS == 8
	  output[output_channel][output_time / NR_TIMES_PER_BLOCK][receiver][polarization][output_time % NR_TIMES_PER_BLOCK] = make_char2(sample.x / 8192, sample.y / 8192);
#endif
	}
      }
#else
      for (unsigned time_minor = 0; time_minor < 16; time_minor ++) {
        unsigned time = time_major + time_minor;

        if (channel < NR_OUTPUT_CHANNELS_PER_SUBBAND * CHANNEL_INTEGRATION_FACTOR) {
          float2 sample = bandPassWeights[channel + 1] * fft_data[time_minor][channel + 1];
sample = make_float2(sample.x / 16, sample.y / 16);
	  //sample = complexMul(sample, v);
          output[receiver][polarization][time][channel] = __float22half2_rn(sample);
        }
      }
#endif
    }

    __syncthreads();
  }
#endif

  //if (infinity_count > 0)
    //printf("%u out of %u infinity\n", infinity_count, total_count);
}


extern "C" __global__ __launch_bounds__(32 * 32)
void postTranspose(
  OutputSample output[NR_OUTPUT_CHANNELS_PER_SUBBAND * CHANNEL_INTEGRATION_FACTOR][NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK][NR_RECEIVERS * NR_POLARIZATIONS][NR_TIMES_PER_BLOCK],
  OutputSample input[NR_RECEIVERS * NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL][NR_OUTPUT_CHANNELS_PER_SUBBAND * CHANNEL_INTEGRATION_FACTOR]
)
{
  __shared__ OutputSample tmp[64][65];

  unsigned recv_pol_major = blockIdx.x * (NR_TIMES_PER_BLOCK < 64 ? 64 / NR_TIMES_PER_BLOCK : 1);
  unsigned time_major     = blockIdx.y * (NR_TIMES_PER_BLOCK < 64 ? NR_TIMES_PER_BLOCK : 64);
  unsigned channel_major  = blockIdx.z * 64;

  for (unsigned y = 0; y < 64; y += 32) {
    for (unsigned x = 0; x < 64; x += 32) {
      unsigned time     = time_major     + (NR_TIMES_PER_BLOCK < 64 ? (y + threadIdx.y) % NR_TIMES_PER_BLOCK : y + threadIdx.y);
      unsigned recv_pol = recv_pol_major + (NR_TIMES_PER_BLOCK < 64 ? (y + threadIdx.y) / NR_TIMES_PER_BLOCK : 0);
      unsigned channel  = channel_major  + x + threadIdx.x;

      if (output_channel_ok(channel) && recv_pol_ok(recv_pol) && output_time_ok(time))
	tmp[y + threadIdx.y][x + threadIdx.x] = input[recv_pol][time][channel];
    }
  }

  __syncthreads();

  for (unsigned y = 0; y < 64; y += 32) {
    for (unsigned x = 0; x < 64; x += 32) {
      unsigned time     = time_major     + (NR_TIMES_PER_BLOCK < 64 ? (x + threadIdx.x) % NR_TIMES_PER_BLOCK : x + threadIdx.x);
      unsigned recv_pol = recv_pol_major + (NR_TIMES_PER_BLOCK < 64 ? (x + threadIdx.x) / NR_TIMES_PER_BLOCK : 0);
      unsigned channel  = channel_major  + y + threadIdx.y;

      if (output_channel_ok(channel) && recv_pol_ok(recv_pol) && output_time_ok(time))
	output[channel][time / NR_TIMES_PER_BLOCK][recv_pol][time % NR_TIMES_PER_BLOCK] = tmp[x + threadIdx.x][y + threadIdx.y];
    }
  }
}


#if defined TEST
// FIXME
// nvcc -DTEST -DNR_RECEIVERS=2 -DNR_BITS=16 -DNR_CHANNELS_PER_SUBBAND=64 -DNR_OUTPUT_CHANNELS_PER_SUBBAND=63 -DNR_SAMPLES_PER_CHANNEL=32 -DNR_POLARIZATIONS=2 -DNR_TAPS=16 -std=c++17 -arch=sm_87 -I/home/romein/packages/nvidia-mathdx-22.11.0-Linux/nvidia/mathdx/22.11/include -I. Correlator/Kernels/FilterAndCorrect.cu
		      //(ps.nrStations() * ps.nrPolarizations() + nrRecvPolPerBlock - 1) / nrRecvPolPerBlock, 1, (ps.nrChannelsPerSubband() + 31) / 32,

#include <iostream>

std::ostream &operator << (std::ostream &str, OutputSample sample)
{
  return str << '(' << (float) sample.x << ", " << (float) sample.y << ')';
}


inline void checkCudaCall(hipError_t error)
{
  if (error != hipSuccess) {
    std::cerr << "error " << error << std::endl;
    exit(1);
  }
}


#if 0
inline __host__ bool operator != (const OutputSample &a, const OutputSample &b)
{
  return a.x != b.x || a.y != b.y;
}
#endif


static const float constWeights[] = {
//#include "../../weights.txt"
#include "weights.txt"
};

#include <string>
#include <stdexcept>

int main()
{
  InputSample  (*input)[NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1][NR_CHANNELS_PER_SUBBAND][NR_RECEIVERS][NR_POLARIZATIONS];
  InputSample  (*transposedInput)[NR_RECEIVERS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1][NR_CHANNELS_PER_SUBBAND];
  OutputSample (*output)[NR_CHANNELS_PER_SUBBAND][NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_BLOCK][NR_RECEIVERS][NR_POLARIZATIONS][NR_TIMES_PER_BLOCK];
  float        (*filterWeights)[NR_CHANNELS_PER_SUBBAND][NR_TAPS];
  float        (*delays)[NR_RECEIVERS][NR_POLARIZATIONS]; // in seconds
  float        (*bandPassWeights)[NR_CHANNELS_PER_SUBBAND];

  checkCudaCall(hipMallocManaged(&output, sizeof(*output)));
  checkCudaCall(hipMallocManaged(&input, sizeof(*input)));
  checkCudaCall(hipMallocManaged(&transposedInput, sizeof(*transposedInput)));
  checkCudaCall(hipMallocManaged(&filterWeights, sizeof(*filterWeights)));
  checkCudaCall(hipMallocManaged(&delays, sizeof(*delays)));
  checkCudaCall(hipMallocManaged(&bandPassWeights, sizeof(*bandPassWeights)));

#if 0
  (*filterWeights)[0][15] = 1.0f;
#elif 1
for (unsigned channel = 0; channel < NR_CHANNELS_PER_SUBBAND; channel ++)
  (*filterWeights)[channel][0] = 1;
#else
  memcpy(filterWeights, constWeights, sizeof constWeights);
#endif

  for (float &bandPassWeight : (*bandPassWeights))
    bandPassWeight = 1.0f;

#if 0
#if NR_BITS == 16
  (*input)[22][4][1][0] = make_short2(128, 0);
#elif NR_BITS == 8
  (*input)[22][4][1][0] = make_char2(127, 0);
#endif
#elif 0
  for (unsigned time = 0; time < NR_SAMPLES_PER_CHANNEL * NR_CHANNELS_PER_SUBBAND; time ++) {
    float phi = 2 * HIP_PI_F * time * 6 / NR_SAMPLES_PER_CHANNEL;
#if NR_BITS == 16
    (*input)[time / NR_CHANNELS_PER_SUBBAND][time % NR_CHANNELS_PER_SUBBAND][1][0] = make_short2((short) (128 * cos(phi)), (short) (128 * sin(phi)));
#elif NR_BITS == 8
    (*input)[time / NR_CHANNELS_PER_SUBBAND][time % NR_CHANNELS_PER_SUBBAND][1][0] = make_char2((short) (127 * cos(phi)), (short) (127 * sin(phi)));
#endif
  }
#else
  for (unsigned channel = 0; channel < NR_CHANNELS_PER_SUBBAND; channel ++) {
    float phi = 2 * HIP_PI_F * 5 * channel / NR_CHANNELS_PER_SUBBAND;
#if NR_BITS == 16
    (*input)[22][channel][1][0] = make_short2((short) roundf(32 * cos(phi)), (short) roundf(32 * sin(phi)));
#elif NR_BITS == 8
    (*input)[22][channel][1][0] = make_char2((short) roundf(32 * cos(phi)), (short) roundf(32 * sin(phi)));
#endif
  }
#endif

  transpose<<<
    dim3((((NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS_PER_SUBBAND) + 31) / 32, (NR_RECEIVERS * NR_POLARIZATIONS + 31) / 32),
    dim3(32, 32)
  >>>(
    * (InputSample (*)[NR_RECEIVERS * NR_POLARIZATIONS][(NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS_PER_SUBBAND]) transposedInput,
    * (InputSample (*)[(NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS_PER_SUBBAND][NR_RECEIVERS * NR_POLARIZATIONS]) input
  );

  filterAndCorrect<<<
    dim3(NR_POLARIZATIONS, NR_RECEIVERS),
    dim3(NR_CHANNELS_PER_SUBBAND / 16, 16)
  >>>(
    *output,
    *transposedInput,
    *filterWeights,
    *delays,
    *bandPassWeights
    /*, 60e6f*/
  );

  checkCudaCall(hipDeviceSynchronize());

#if 1
  std::cout << "testing ..." << std::endl;

  for (unsigned receiver = 0; receiver < NR_RECEIVERS; receiver ++)
    for (unsigned polarization = 0; polarization < NR_POLARIZATIONS; polarization ++)
      for (unsigned time = 0; time < NR_SAMPLES_PER_CHANNEL; time ++)
	for (unsigned channel = 0; channel < NR_CHANNELS_PER_SUBBAND; channel ++) {
	  OutputSample &sample = (*output)[channel][time / NR_TIMES_PER_BLOCK][receiver][polarization][time % NR_TIMES_PER_BLOCK];

	  if ((int) sample.x != 0 || (int) sample.y != 0)
	    std::cout << "output[" << channel << "][" << time / NR_TIMES_PER_BLOCK << "][" << receiver << "][" << polarization << "][" << time % NR_TIMES_PER_BLOCK << "] = " << sample << std::endl;
	}
#endif

  checkCudaCall(hipFree(bandPassWeights));
  checkCudaCall(hipFree(delays));
  checkCudaCall(hipFree(filterWeights));
  checkCudaCall(hipFree(transposedInput));
  checkCudaCall(hipFree(input));
  checkCudaCall(hipFree(output));
}
#endif
