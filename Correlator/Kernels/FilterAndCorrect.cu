#include "hip/hip_runtime.h"
#include <hip/hip_fp8.h>
#include <hip/hip_fp16.h>
#include <hip/hip_math_constants.h>
#include <cufftdx.hpp>

#if defined __CUDA_ARCH__
#include <sm_61_intrinsics.hpp> // must include this explicitly for __dp4a when compiling with NVRTC
#endif


//#define SUBBAND_BANDWIDTH 195312.5

#define REAL	0
#define IMAG	1
#define COMPLEX	2

#define ALIGN(N, A) (((N) + (A) - 1) / (A) * (A))
#define MIN(A,B) ((A)<(B)?(A):(B))

inline __device__ void prefetch(const void *ptr, unsigned size, unsigned tid, unsigned nr_threads)
{
#if __CUDA_ARCH__ >= 900
  if (tid == 0)
    asm ("cp.async.bulk.prefetch.L2.global [%0],%1;" :: "l" (ptr), "r" (size));
#else
  for (unsigned i = 0; i < size; i += 8 * nr_threads)
    asm ("prefetch.global.L2 [%0];" :: "l" (static_cast<const char *>(ptr) + i + 8 * tid));
#endif
}


typedef float2 fcomplex; // std::complex not properly supported yet

__device__ inline fcomplex operator + (fcomplex a, fcomplex b)
{
  return make_float2(a.x + b.x, a.y + b.y);
}

__device__ inline fcomplex operator - (fcomplex a, fcomplex b)
{
  return make_float2(a.x - b.x, a.y - b.y);
}

__device__ inline fcomplex operator * (float a, fcomplex b)
{
  return make_float2(a * b.x, a * b.y);
}

__device__ inline fcomplex operator *= (fcomplex &a, float b)
{
  return make_float2(a.x *= b, a.y *= b);
}

#if 0
__device__ inline __half2 operator * (__half2 a, __half2 b)
{
  return make_half2(a.x * b.x - a.y * b.y, a.x * b.y - a.y * b.x); // TODO: use vector intrinsics
}
#endif

__device__ inline fcomplex operator += (fcomplex &a, fcomplex b)
{
  return make_float2(a.x += b.x, a.y += b.y);
}


#define complexMul(a,b) make_float2((a).x * (b).x - (a).y * (b).y, (a).y * (b).x + (a).x * (b).y)


#if INPUT_SAMPLE_FORMAT == I16
typedef short2  InputSample;
#elif INPUT_SAMPLE_FORMAT == I8
typedef char2 InputSample;
#else
#error unsupport input sample format
#endif

#if OUTPUT_SAMPLE_FORMAT == FP16
typedef __half2 OutputSample;
#elif OUTPUT_SAMPLE_FORMAT == E4M3
typedef __hip_fp8x2_e4m3_fnuz OutputSample;
#elif OUTPUT_SAMPLE_FORMAT == E5M2
typedef __hip_fp8x2_e5m2_fnuz OutputSample;
#elif OUTPUT_SAMPLE_FORMAT == I8
typedef char2 OutputSample;
#else
#error unsupported output sample format
#endif


#if FIR_FILTER_SAMPLE_FORMAT == FP32

template <typename T> __device__ inline float2 sampleToComplexFloat(T sample)
{
  return make_float2(sample.x, sample.y);
}

#elif FIR_FILTER_SAMPLE_FORMAT == FP16

template <typename T> __device__ inline __half2 sampleToComplexFloat(T sample)
{
  return make_half2(sample.x, sample.y);
}

#endif

#define NR_BITS(FORMAT) (FORMAT == FP32 || FORMAT == I32 ? 32 : \
			 FORMAT == FP16 || FORMAT == BF16 || FORMAT == I16 ? 16 : \
			 FORMAT == E4M3 || FORMAT == E5M2 || FORMAT ==  I8 ?  8 : \
			 FORMAT == I4 ? 4 : 0)
#define NR_TIMES_PER_OUTPUT_BLOCK      	(128 / NR_BITS(OUTPUT_SAMPLE_FORMAT))


#if defined INPUT_CUSTOM_CODE
INPUT_CUSTOM_CODE
#elif defined RING_BUFFER_SIZE

typedef InputSample InputType[NR_RECEIVERS][NR_POLARIZATIONS][RING_BUFFER_SIZE];

InputSample readSample(const InputType input, unsigned receiver, unsigned polarization, unsigned time, unsigned channel, unsigned ringBufferStartIndex)
{
  return input[receiver][polarization][(time * NR_CHANNELS + channel + ringBufferStartIndex) % RING_BUFFER_SIZE];
}

#else

typedef InputSample InputType[NR_RECEIVERS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1][NR_CHANNELS];

//InputSample readSample(InputType input, unsigned receiver, unsigned polarization, unsigned time, unsigned channel)
//{
//  return input[receiver][polarization][time][channel];
//}

#endif


#define NR_THREADS ((NR_CHANNELS + NR_CHANNELS_PER_THREAD - 1) / NR_CHANNELS_PER_THREAD * NR_TIMES_PER_ITERATION)

extern "C" __global__ __launch_bounds__(NR_THREADS /*, 3 */)
void filterAndCorrect(
  OutputSample      output[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_OUTPUT_BLOCK][NR_RECEIVERS][NR_POLARIZATIONS][NR_TIMES_PER_OUTPUT_BLOCK],
  const InputType   input,
  const float       filterWeights[NR_TAPS][NR_CHANNELS]
#if defined APPLY_DELAYS
  , const float     delays[NR_RECEIVERS][NR_POLARIZATIONS] // in seconds
#endif
#if defined APPLY_BANDPASS_WEIGHTS
  , const float     bandPassWeights[NR_CHANNELS]
#endif
#if defined RING_BUFFER_SIZE
  , unsigned        ringBufferStartIndex
#endif
  //double	    subbandFrequency
)
{
#if defined __CUDA_ARCH__
  unsigned tid = threadIdx.y * blockDim.x + threadIdx.x;

#if 0
  unsigned polarization = blockIdx.x;
  unsigned receiver     = blockIdx.y;
  unsigned time_major   = blockIdx.z * NR_TIMES_PER_ITERATION;
#else
#if __CUDA_ARCH__ == 870
  constexpr unsigned divisor = 1;
#else
  constexpr unsigned divisor = MIN(8, NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_ITERATION);
#endif

  unsigned polarization = (blockIdx.x / divisor) % NR_POLARIZATIONS;
  unsigned receiver     = (blockIdx.x / (divisor * NR_POLARIZATIONS)) % NR_RECEIVERS;
  unsigned time_major   = (blockIdx.x % divisor + (blockIdx.x / (divisor * NR_POLARIZATIONS * NR_RECEIVERS)) * divisor) * NR_TIMES_PER_ITERATION;
#endif

  //prefetch(input[receiver][polarization][time_major], NR_CHANNELS * (NR_TAPS - 1 + NR_TIMES_PER_ITERATION) * sizeof(InputSample), tid, NR_THREADS);

#if FFT_SAMPLE_FORMAT != FP32
#error unspported FFT precision
#endif

  using namespace cufftdx;
  using FFT = decltype(Block() +
	      Size<NR_CHANNELS>() +
	      Type<fft_type::c2c>() +
	      Direction<fft_direction::forward>() +
	      Precision<float>() +
	      ElementsPerThread<NR_CHANNELS_PER_THREAD>() +
	      FFTsPerBlock<NR_TIMES_PER_ITERATION>() +
	      //BlockDim<NR_CHANNELS, 1, 1>() + // Not yet supported by cuFFTDx
	      SM<__CUDA_ARCH__>());

  __shared__ union {
    FFT::value_type shared_mem[FFT::shared_memory_size / sizeof(FFT::value_type)];
    float2 fft_data[NR_TIMES_PER_ITERATION][NR_CHANNELS | 1];
  } u;

#if defined APPLY_DELAYS
  float  phi = -2 * HIP_PI_F * delays[receiver][polarization];
  float2 v   = make_float2(cosf(phi), sinf(phi));
#endif

  FFT::value_type thread_data[FFT::storage_size];
  using complex_type = typename FFT::value_type;

#pragma unroll
  for (unsigned channel_major = 0, channel; (channel = channel_major + tid) < NR_CHANNELS; channel_major += NR_THREADS) {
    float2 cachedSamples[NR_TIMES_PER_ITERATION + NR_TAPS - 1];

    for (unsigned i = 0; i < NR_TIMES_PER_ITERATION + NR_TAPS - 1; i ++)
#if defined INPUT_CUSTOM_CODE
      cachedSamples[i] = sampleToComplexFloat(readSample(input, receiver, polarization, time_major + i, channel));
#elif RING_BUFFER_SIZE
      cachedSamples[i] = sampleToComplexFloat(readSample(input, receiver, polarization, time_major + i, channel, ringBufferStartIndex));
#else
      cachedSamples[i] = sampleToComplexFloat(input[receiver][polarization][time_major + i][channel]);
#endif

#pragma unroll
    for (unsigned time_minor = 0; time_minor < NR_TIMES_PER_ITERATION; time_minor ++)  {
#if FIR_FILTER_SAMPLE_FORMAT == FP32
      float2 sum = make_float2(0, 0);

#pragma unroll
      for (unsigned tap = 0; tap < NR_TAPS; tap ++)
	sum += filterWeights[NR_TAPS - 1 - tap][channel] * cachedSamples[time_minor + tap];

      u.fft_data[time_minor][channel] = sum;
      //thread_data[channel_idx] = FFT::value_type(sum.x, sum.y);
#elif FIR_FILTER_SAMPLE_FORMAT == FP16
      __half2 sum = make_half2(0, 0);

#pragma unroll
      for (unsigned tap = 0; tap < NR_TAPS; tap ++)
	sum = __hfma2(__float2half2_rn(filterWeights[NR_TAPS - 1 - tap][channel] /* / 16 */), sampleToComplexFloat(input[receiver][polarization][time_major + time_minor + tap][channel]), sum);

      u.fft_data[time_minor][channel] = __half22float2(sum);
#elif FIR_FILTER_SAMPLE_FORMAT == I8
      int sum[COMPLEX] = {0};

	for (unsigned ri = 0; ri < COMPLEX; ri ++) {
#if 0
	for (int tap = NR_TAPS / 4; -- tap >= 0;) {
	  sum[ri] = __dp4a(history[ri].i[tap], _weights.i[tap], sum[ri]);
	  history[ri].i[tap] = __funnelshift_l(history[ri].i[tap], history[ri].i[tap - 1], 8);
	}
#else
	for (unsigned tap = 0; tap < NR_TAPS; tap ++)
	  sum[ri] += (int) history[ri].ch[tap] * roundf(127 * filterWeights[tap][channel]);

	for (unsigned tap = NR_TAPS - 1; tap > 0; tap --)
	  history[ri].ch[tap] = history[ri].ch[tap - 1];
#endif
      }

      u.fft_data[time_minor][channel] = make_float2(sum[REAL], sum[IMAG]);
#endif
    }
  }

  __syncthreads();

  unsigned stride = size_of<FFT>::value / FFT::elements_per_thread;

  for (unsigned i = 0; i < FFT::elements_per_thread; ++ i)
    thread_data[i] = * ((FFT::value_type *) &u.fft_data[threadIdx.y][i * stride + threadIdx.x]);

  FFT().execute(thread_data, u.shared_mem);

  for (unsigned i = 0; i < FFT::elements_per_thread; ++ i)
    * ((complex_type *) &u.fft_data[threadIdx.y][i * stride + threadIdx.x]) = thread_data[i];

  __syncthreads();

  {
    for (unsigned i = 0; i < NR_CHANNELS * NR_TIMES_PER_ITERATION; i += NR_THREADS) {
      unsigned channel    = (i + tid) / NR_TIMES_PER_ITERATION;
      unsigned time_minor = (i + tid) % NR_TIMES_PER_ITERATION;
      unsigned time       = time_major + time_minor;

      /*if (NR_CHANNELS % 16 == 0 || channel < NR_CHANNELS)*/ {
	//double frequency = subbandFrequency - .5 * SUBBAND_BANDWIDTH + channel * (SUBBAND_BANDWIDTH / NR_CHANNELS);
	float2 sample = u.fft_data[time_minor][channel]; // TODO: use __half2 ???
#if defined APPLY_BANDPASS_WEIGHTS
	sample *= bandPassWeights[channel];
#endif
#if defined APPLY_DELAYS
	sample = complexMul(sample, v);
#endif
#if defined OUTPUT_SCALE_FACTOR
	sample *= OUTPUT_SCALE_FACTOR;
#endif
#if OUTPUT_SAMPLE_FORMAT == FP16
	OutputSample outputSample = __float22half2_rn(sample);
#elif OUTPUT_SAMPLE_FORMAT == E4M3
	OutputSample outputSample = __hip_fp8x2_e4m3_fnuz(sample);
#elif OUTPUT_SAMPLE_FORMAT == E5M2
	OutputSample outputSample = __hip_fp8x2_e5m2_fnuz(sample);
#elif OUTPUT_SAMPLE_FORMAT == I8
	sample = make_float2(fmaxf(sample.x, -127.f), fmaxf(sample.y, -127.f)); // TCC cannot handle -128 as a calue
	sample = make_float2(fminf(sample.x,  127.f), fminf(sample.y,  127.f));
	sample = make_float2(rintf(sample.x), rintf(sample.y));

	OutputSample outputSample = make_char2(sample.x, sample.y);
#endif
	output[channel][time / NR_TIMES_PER_OUTPUT_BLOCK][receiver][polarization][time % NR_TIMES_PER_OUTPUT_BLOCK] = outputSample;
      }
    }
  }
#endif
}


#if defined TEST
#warning FIXME
// nvcc -DTEST -DNR_RECEIVERS=2 -DNR_BITS=16 -DNR_CHANNELS=64 -DNR_SAMPLES_PER_CHANNEL=32 -DNR_POLARIZATIONS=2 -DNR_TAPS=16 -std=c++17 -arch=sm_87 -I/home/romein/packages/nvidia-mathdx-22.11.0-Linux/nvidia/mathdx/22.11/include -I. Correlator/Kernels/FilterAndCorrect.cu

#include <iostream>

std::ostream &operator << (std::ostream &str, OutputSample sample)
{
  return str << '(' << (float) sample.x << ", " << (float) sample.y << ')';
}


inline void checkCudaCall(hipError_t error)
{
  if (error != hipSuccess) {
    std::cerr << "error " << error << std::endl;
    exit(1);
  }
}


#if 0
inline __host__ bool operator != (const OutputSample &a, const OutputSample &b)
{
  return a.x != b.x || a.y != b.y;
}
#endif


static const float constWeights[] = {
//#include "../../weights.txt"
#include "weights.txt"
};

#include <string>
#include <stdexcept>

int main()
{
  InputSample  (*input)[NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1][NR_CHANNELS][NR_RECEIVERS][NR_POLARIZATIONS];
  InputSample  (*transposedInput)[NR_RECEIVERS][NR_POLARIZATIONS][NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1][NR_CHANNELS];
  OutputSample (*output)[NR_CHANNELS][NR_SAMPLES_PER_CHANNEL / NR_TIMES_PER_OUTPUT_BLOCK][NR_RECEIVERS][NR_POLARIZATIONS][NR_TIMES_PER_OUTPUT_BLOCK];
  float        (*filterWeights)[NR_TAPS][NR_CHANNELS];
  float        (*delays)[NR_RECEIVERS][NR_POLARIZATIONS]; // in seconds
  float        (*bandPassWeights)[NR_CHANNELS];

  checkCudaCall(hipMallocManaged(&output, sizeof(*output)));
  checkCudaCall(hipMallocManaged(&input, sizeof(*input)));
  checkCudaCall(hipMallocManaged(&transposedInput, sizeof(*transposedInput)));
  checkCudaCall(hipMallocManaged(&filterWeights, sizeof(*filterWeights)));
  checkCudaCall(hipMallocManaged(&delays, sizeof(*delays)));
  checkCudaCall(hipMallocManaged(&bandPassWeights, sizeof(*bandPassWeights)));

#if 0
  (*filterWeights)[15][0] = 1.0f;
#elif 1
for (unsigned channel = 0; channel < NR_CHANNELS; channel ++)
  (*filterWeights)[0][channel] = 1;
#else
  memcpy(filterWeights, constWeights, sizeof constWeights);
#endif

  for (float &bandPassWeight : (*bandPassWeights))
    bandPassWeight = 1.0f;

#if 0
#if NR_BITS == 16
  (*input)[22][4][1][0] = make_short2(128, 0);
#elif NR_BITS == 8
  (*input)[22][4][1][0] = make_char2(127, 0);
#endif
#elif 0
  for (unsigned time = 0; time < NR_SAMPLES_PER_CHANNEL * NR_CHANNELS; time ++) {
    float phi = 2 * HIP_PI_F * time * 6 / NR_SAMPLES_PER_CHANNEL;
#if NR_BITS == 16
    (*input)[time / NR_CHANNELS][time % NR_CHANNELS][1][0] = make_short2((short) (128 * cos(phi)), (short) (128 * sin(phi)));
#elif NR_BITS == 8
    (*input)[time / NR_CHANNELS][time % NR_CHANNELS][1][0] = make_char2((short) (127 * cos(phi)), (short) (127 * sin(phi)));
#endif
  }
#else
  for (unsigned channel = 0; channel < NR_CHANNELS; channel ++) {
    float phi = 2 * HIP_PI_F * 5 * channel / NR_CHANNELS;
#if NR_BITS == 16
    (*input)[22][channel][1][0] = make_short2((short) roundf(32 * cos(phi)), (short) roundf(32 * sin(phi)));
#elif NR_BITS == 8
    (*input)[22][channel][1][0] = make_char2((short) roundf(32 * cos(phi)), (short) roundf(32 * sin(phi)));
#endif
  }
#endif

  transpose<<<
    dim3((((NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS) + 31) / 32, (NR_RECEIVERS * NR_POLARIZATIONS + 31) / 32),
    dim3(32, 32)
  >>>(
    * (InputSample (*)[NR_RECEIVERS * NR_POLARIZATIONS][(NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS]) transposedInput,
    * (InputSample (*)[(NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS][NR_RECEIVERS * NR_POLARIZATIONS]) input
  );

  filterAndCorrect<<<
    dim3(NR_POLARIZATIONS, NR_RECEIVERS),
    dim3(NR_CHANNELS / 16, 16)
  >>>(
    *output,
    *transposedInput,
    *filterWeights,
    *delays,
    *bandPassWeights
    /*, 60e6f*/
  );

  checkCudaCall(hipDeviceSynchronize());

#if 1
  std::cout << "testing ..." << std::endl;

  for (unsigned receiver = 0; receiver < NR_RECEIVERS; receiver ++)
    for (unsigned polarization = 0; polarization < NR_POLARIZATIONS; polarization ++)
      for (unsigned time = 0; time < NR_SAMPLES_PER_CHANNEL; time ++)
	for (unsigned channel = 0; channel < NR_CHANNELS; channel ++) {
	  OutputSample &sample = (*output)[channel][time / NR_TIMES_PER_OUTPUT_BLOCK][receiver][polarization][time % NR_TIMES_PER_OUTPUT_BLOCK];

	  if ((int) sample.x != 0 || (int) sample.y != 0)
	    std::cout << "output[" << channel << "][" << time / NR_TIMES_PER_OUTPUT_BLOCK << "][" << receiver << "][" << polarization << "][" << time % NR_TIMES_PER_OUTPUT_BLOCK << "] = " << sample << std::endl;
	}
#endif

  checkCudaCall(hipFree(bandPassWeights));
  checkCudaCall(hipFree(delays));
  checkCudaCall(hipFree(filterWeights));
  checkCudaCall(hipFree(transposedInput));
  checkCudaCall(hipFree(input));
  checkCudaCall(hipFree(output));
}
#endif
