#include "hip/hip_runtime.h"
#include <hip/hip_fp16.h>



#define REAL	0
#define IMAG	1
#define COMPLEX	2


#if NR_BITS == 16
typedef short2  InputSample;
typedef __half2 OutputSample;
#elif NR_BITS == 8
typedef char2 InputSample, OutputSample;
#else
#error unsupport NR_BITS
#endif


#define NR_TIMES_PER_BLOCK      	(128 / (NR_BITS))
#define CHANNEL_INTEGRATION_FACTOR	(NR_CHANNELS_PER_SUBBAND == 1 ? 1 : (NR_CHANNELS_PER_SUBBAND - 1) / NR_OUTPUT_CHANNELS_PER_SUBBAND)
#define NR_OUTPUT_SAMPLES_PER_CHANNEL	(CHANNEL_INTEGRATION_FACTOR * NR_SAMPLES_PER_CHANNEL)


__device__ inline bool time_ok(unsigned time)
{
  return (NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS_PER_SUBBAND % 64 == 0 || time < (NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS_PER_SUBBAND;
}


__device__ inline bool recv_pol_ok(unsigned recv_pol)
{
  return NR_RECEIVERS * NR_POLARIZATIONS % 64 == 0 || recv_pol < NR_RECEIVERS * NR_POLARIZATIONS;
}


extern "C" __global__ __launch_bounds__(32 * 32)
void transpose(
  InputSample output[NR_RECEIVERS * NR_POLARIZATIONS][(NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS_PER_SUBBAND],
#if defined HAS_INTEGRATED_MEMORY && !defined TEST
  const InputSample input[NR_RING_BUFFER_SAMPLES_PER_SUBBAND][NR_RECEIVERS * NR_POLARIZATIONS],
  unsigned startIndex
#else
  const InputSample input[(NR_SAMPLES_PER_CHANNEL + NR_TAPS - 1) * NR_CHANNELS_PER_SUBBAND][NR_RECEIVERS * NR_POLARIZATIONS]
#endif
)
{
  __shared__ InputSample tmp[64][64 + 1 /* one wider, to avoid bank conflicts */];

  unsigned recv_pol_major = blockIdx.x * 64;
  unsigned time_major     = blockIdx.y * 64;

  for (unsigned y = 0; y < 64; y += 32) {
    for (unsigned x = 0; x < 64; x += 32) {
      unsigned time     = time_major     + y + threadIdx.y;
      unsigned recv_pol = recv_pol_major + x + threadIdx.x;

      if (time_ok(time) && recv_pol_ok(recv_pol))
#if defined HAS_INTEGRATED_MEMORY && !defined TEST
	tmp[y + threadIdx.y][x + threadIdx.x] = input[(time + startIndex) % NR_RING_BUFFER_SAMPLES_PER_SUBBAND][recv_pol];
#else
	tmp[y + threadIdx.y][x + threadIdx.x] = input[time][recv_pol];
#endif
    }
  }

  __syncthreads();

  for (unsigned y = 0; y < 64; y += 32) {
    for (unsigned x = 0; x < 64; x += 32) {
      unsigned time     = time_major     + x + threadIdx.x;
      unsigned recv_pol = recv_pol_major + y + threadIdx.y;

      if (time_ok(time) && recv_pol_ok(recv_pol))
	output[recv_pol][time] = tmp[x + threadIdx.x][y + threadIdx.y];
    }
  }
}
